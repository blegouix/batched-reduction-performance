// Copyright (C) The DDC development team, see COPYRIGHT.md file
//
// SPDX-License-Identifier: MIT

#include <cub/block/block_reduce.cuh>
#include <cuda/std/mdspan>

#include <benchmark/benchmark.h>

#include <batched_reduction_performance/batched_reduction_performance.hpp>

static constexpr std::size_t BlockDim1D = 64;
static constexpr std::size_t BlockDim2D_1 = 4;
static constexpr std::size_t BlockDim2D_2 = 4;

template <std::size_t M, std::size_t N, class BatchedReductionOperator,
          class Layout>
class BatchedReductionBenchmark {
  static_assert(M >= BlockDim1D &&
                "M has to be equal or greater than BlockDim1D");
  static_assert(M >= BlockDim2D_1 * BlockDim2D_2 &&
                "M has to be equal or greater than BlockDim2D_1*BlockDim2D_2");
  static_assert(
      N <= 1024 &&
      "N has to be at most 1024 which is the number of threads per SM");

public:
  static void run(benchmark::State &state) {
    double *data_in_ptr = nullptr;
    hipMalloc(&data_in_ptr, M * N * sizeof(double));

    cuda::std::mdspan<double, cuda::std::extents<std::size_t, M, N>, Layout>
        data_in = wrapper::wrap<M, N, Layout>(data_in_ptr);
    filler::fill<BlockDim2D_1, BlockDim2D_2>(data_in);
    // printer::print<BlockDim2D_1, BlockDim2D_2>(data_in);

    double *data_out_ptr = nullptr;
    hipMalloc(&data_out_ptr, M * sizeof(double));

    cuda::std::mdspan<double, cuda::std::extents<std::size_t, M>> data_out(
        data_out_ptr);

    for (auto _ : state) {
      BatchedReductionOperator::run(data_out, data_in);
    }
    state.SetBytesProcessed(int64_t(state.iterations()) *
                            int64_t(M * N * sizeof(double)));

    // printer::print<BlockDim1D>(data_out);
    checker::check<BlockDim1D>(data_out, data_in);

    hipFree(data_in_ptr);
    hipFree(data_out_ptr);
  }
};

#define BENCHMARKS(M, N)                                                         \
  BENCHMARK(BatchedReductionBenchmark<                                           \
            M, N, batched_reduction_operator::Sequential<BlockDim1D>,            \
            cuda::std::layout_right>::run);                                      \
  BENCHMARK(BatchedReductionBenchmark<                                           \
            M, N, batched_reduction_operator::Sequential<BlockDim1D>,            \
            cuda::std::layout_left>::run);                                       \
  BENCHMARK(BatchedReductionBenchmark<                                           \
            M, N, batched_reduction_operator::Sequential<BlockDim1D>,            \
            cuda::std::layout_stride>::run);                                     \
  /*                                                                             \
    BENCHMARK(                                                                   \
        BatchedReductionBenchmark<M, N,                                          \
                                  batched_reduction_operator::CooperativeGroups, \
                                  cuda::std::layout_right>::run);                \
    BENCHMARK(                                                                   \
        BatchedReductionBenchmark<M, N,                                          \
                                  batched_reduction_operator::CooperativeGroups, \
                                  cuda::std::layout_left>::run);                 \
    BENCHMARK(                                                                   \
        BatchedReductionBenchmark<M, N,                                          \
                                  batched_reduction_operator::CooperativeGroups, \
                                  cuda::std::layout_stride>::run);               \
  */                                                                             \
  BENCHMARK(                                                                     \
      BatchedReductionBenchmark<M, N,                                            \
                                batched_reduction_operator::CUBBlockReduction<   \
                                    hipcub::BLOCK_REDUCE_WARP_REDUCTIONS>,          \
                                cuda::std::layout_right>::run);                  \
  BENCHMARK(                                                                     \
      BatchedReductionBenchmark<M, N,                                            \
                                batched_reduction_operator::CUBBlockReduction<   \
                                    hipcub::BLOCK_REDUCE_WARP_REDUCTIONS>,          \
                                cuda::std::layout_left>::run);                   \
  BENCHMARK(                                                                     \
      BatchedReductionBenchmark<M, N,                                            \
                                batched_reduction_operator::CUBBlockReduction<   \
                                    hipcub::BLOCK_REDUCE_WARP_REDUCTIONS>,          \
                                cuda::std::layout_stride>::run);                 \
  BENCHMARK(BatchedReductionBenchmark<                                           \
            M, N,                                                                \
            batched_reduction_operator::CUBBlockReduction<                       \
                hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY>,                      \
            cuda::std::layout_right>::run);                                      \
  BENCHMARK(BatchedReductionBenchmark<                                           \
            M, N,                                                                \
            batched_reduction_operator::CUBBlockReduction<                       \
                hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY>,                      \
            cuda::std::layout_left>::run);                                       \
  BENCHMARK(BatchedReductionBenchmark<                                           \
            M, N,                                                                \
            batched_reduction_operator::CUBBlockReduction<                       \
                hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY>,                      \
            cuda::std::layout_stride>::run);

/*
BENCHMARKS(65536, 64);
BENCHMARKS(65536, 128);
BENCHMARKS(65536, 256);
BENCHMARKS(65536, 512);
BENCHMARKS(65536, 1024);

BENCHMARKS(4096, 1024);
BENCHMARKS(8192, 1024);
BENCHMARKS(16384, 1024);
BENCHMARKS(32768, 1024);
BENCHMARKS(65536, 1024);
*/

BENCHMARKS(64, 1024);
BENCHMARKS(1024, 1024);
BENCHMARKS(65536, 1024);

int main(int argc, char **argv) {
  ::benchmark::Initialize(&argc, argv);
  if (::benchmark::ReportUnrecognizedArguments(argc, argv)) {
    return 1;
  }
  {
    ::benchmark::RunSpecifiedBenchmarks();
  }
  ::benchmark::Shutdown();
  return 0;
}

// Copyright (C) The DDC development team, see COPYRIGHT.md file
//
// SPDX-License-Identifier: MIT

#include <cuda/std/mdspan>

#include <benchmark/benchmark.h>

#include <batched_reduction_performance/batched_reduction_performance.hpp>

// TODO restore .cpp extension

static constexpr std::size_t M = 32;
static constexpr std::size_t N = 1024;

void dummy_benchmark(benchmark::State &state) {
  double *data_in_ptr = nullptr;
  hipMalloc(&data_in_ptr, M * N * sizeof(double));

  cuda::std::mdspan<double, cuda::std::extents<std::size_t, M, N>> data_in(
      data_in_ptr);
  filler::fill(data_in);
  // printer::print(mat);

  double *data_out_ptr = nullptr;
  hipMalloc(&data_out_ptr, N * sizeof(double));

  cuda::std::mdspan<double, cuda::std::extents<std::size_t, N>> data_out(
      data_out_ptr);

  for (auto _ : state) {
    batched_reduction_kernel::Sequential::run(data_out, data_in);
  }
  state.SetBytesProcessed(int64_t(state.iterations()) *
                          int64_t(state.range(0) * sizeof(double)));

  hipFree(data_in_ptr);
  hipFree(data_out_ptr);
}

std::size_t constexpr dummy_param = 128;

BENCHMARK(dummy_benchmark)->Arg(dummy_param);

int main(int argc, char **argv) {
  ::benchmark::Initialize(&argc, argv);
  if (::benchmark::ReportUnrecognizedArguments(argc, argv)) {
    return 1;
  }
  {
    ::benchmark::RunSpecifiedBenchmarks();
  }
  ::benchmark::Shutdown();
  return 0;
}

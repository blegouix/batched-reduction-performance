// Copyright (C) The DDC development team, see COPYRIGHT.md file
//
// SPDX-License-Identifier: MIT

#include <cuda/std/mdspan>

#include <benchmark/benchmark.h>

#include <batched_reduction_performance/batched_reduction_performance.hpp>

// TODO restore .cpp extension

static constexpr std::size_t M = 32;
static constexpr std::size_t N = 1024;

void dummy_benchmark(benchmark::State &state) {
  double *mat_ptr = nullptr;
  hipMalloc(&mat_ptr, M * N * sizeof(double));

  cuda::std::mdspan<double, cuda::std::extents<std::size_t, M, N>> mat(mat_ptr);
  filler::fill(mat);

  for (auto _ : state) {
    batched_reduction_kernel::dummy_kernel();
  }
  state.SetBytesProcessed(int64_t(state.iterations()) *
                          int64_t(state.range(0) * sizeof(double)));

  hipFree(mat_ptr);
}

std::size_t constexpr dummy_param = 128;

BENCHMARK(dummy_benchmark)->Arg(dummy_param);

int main(int argc, char **argv) {
  ::benchmark::Initialize(&argc, argv);
  if (::benchmark::ReportUnrecognizedArguments(argc, argv)) {
    return 1;
  }
  {
    ::benchmark::RunSpecifiedBenchmarks();
  }
  ::benchmark::Shutdown();
  return 0;
}
